#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>
#include "../OptixLaunchParameters.h"
#include "../OptixData.h"

#define EPSILON 0.000001f

extern "C" {
__constant__ OptixLaunchParameters optixLaunchParameters;
}

extern "C" __global__ void __raygen__rg()
{
    const unsigned int idx = optixGetLaunchIndex().x;

    const RayGenData* rayGenData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
//    printf("RayGen %d\tOrigin: (%.2f,%.2f,%.2f)\tDirection: (%.2f,%.2f,%.2f)\n", idx,
//           rayGenData->origins[idx].x, rayGenData->origins[idx].y, rayGenData->origins[idx].z,
//           rayGenData->directions[idx].x, rayGenData->directions[idx].y, rayGenData->directions[idx].z);
    optixTrace(
            optixLaunchParameters.handle,
            rayGenData->origins[idx],
            rayGenData->directions[idx],
            0,                      // tmin
            1 + EPSILON,            // tmax
            0.0f,                 // Ignored and removed by the compiler if motion is not enabled
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, // No need to disable anyhit if this is done in the acceleration structures
            0,                   // SBT offset, only used for different ray types
            0,                   // SBT stride, only used for different ray types
            0);               // missSBTIndex
}

extern "C" __global__ void __miss__ms()
{

}


extern "C" __global__ void __closesthit__ch()
{
    const unsigned int idx = optixGetLaunchIndex().x;

//    printf("Hit %d: %.5f\n", idx, optixGetRayTmax());
    HitGroupData* hitGroupData = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());
    hitGroupData->hit = true;
}