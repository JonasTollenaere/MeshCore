#include "hip/hip_runtime.h"
#include <glm/gtc/type_ptr.hpp>
#include "CudaWorldSpaceMesh.h"
#include "CudaRay.h"
#include "../cuda/CudaTriangleTriangleIntersect.h"

#define BLOCK_SIZE 32

CudaWorldSpaceMesh::CudaWorldSpaceMesh(const WorldSpaceMesh &worldSpaceMesh, const CudaStream& cudaStream):
    modelSpaceVertices(nullptr),
    worldSpaceVertices(nullptr),
    triangles(nullptr),
    transformation(nullptr),
    flags(nullptr),
    cudaStream(&cudaStream.getStream())
{
    // Allocate and the transformation
    Transformation transform(worldSpaceMesh.getModelTransformation());
    hipMalloc(&this->transformation, sizeof(Transformation));

    // Allocate and copy the modelSpaceVertices
    this->numberOfVertices = worldSpaceMesh.getModelSpaceMesh().getVertices().size();
    unsigned int verticesBytes = this->numberOfVertices * sizeof(Vertex);
    hipMalloc(&this->modelSpaceVertices, verticesBytes);
    hipMemcpyAsync(this->modelSpaceVertices, worldSpaceMesh.getModelSpaceMesh().getVertices().data(), verticesBytes, hipMemcpyHostToDevice, *this->cudaStream);

    // Allocate and update the transformation, worldSpaceVertices
    hipMalloc(&this->worldSpaceVertices, verticesBytes);
    this->setModelTransformation(worldSpaceMesh.getModelTransformation());

    // Allocate and copy the triangles
    this->numberOfTriangles = worldSpaceMesh.getModelSpaceMesh().getTriangles().size();
    unsigned int triangleBytes = this->numberOfTriangles * sizeof(Triangle);
    hipMalloc(&this->triangles, triangleBytes);
    hipMemcpyAsync(this->triangles, worldSpaceMesh.getModelSpaceMesh().getTriangles().data(), triangleBytes, hipMemcpyHostToDevice, *this->cudaStream);

    // Allocate flags
    hipMalloc(&this->flags, 2 * sizeof(bool));

}

CudaWorldSpaceMesh::~CudaWorldSpaceMesh() {
    hipFree(this->transformation);
    hipFree(this->modelSpaceVertices);
    hipFree(this->triangles);
    hipFree(this->worldSpaceVertices);
    hipFree(this->flags);
}

__global__
void calculateWorldSpaceVertices(const Transformation* transformation, const Vertex* modelSpaceVertices, Vertex* worldSpaceVertices, const unsigned numberOfVertices)
{
    unsigned int vertexId = threadIdx.x + blockIdx.x * blockDim.x;
    if(vertexId < numberOfVertices){
        worldSpaceVertices[vertexId] = Vertex(*transformation * glm::vec4(modelSpaceVertices[vertexId], 1));
    }
}

void CudaWorldSpaceMesh::setModelTransformation(const Transformation &transformation) {
    hipMemcpyAsync(this->transformation, &transformation, sizeof(Transformation), hipMemcpyHostToDevice, *this->cudaStream);
    unsigned blockSize = BLOCK_SIZE;
    unsigned int nBlocks= this->numberOfVertices / blockSize + (this->numberOfVertices % blockSize == 0 ? 0 : 1);
    calculateWorldSpaceVertices <<< nBlocks, blockSize, 0, *this->cudaStream >>> (this->transformation, this->modelSpaceVertices, this->worldSpaceVertices, this->numberOfVertices);
}


__global__
void rayTriangleInsideKernel(const Triangle* innerTriangles, const Vertex* innerVertices, const unsigned numberOfInnerVertices, const unsigned int numberOfInnerTriangles,
                             const Triangle* outerTriangles, const Vertex* outerVertices, const unsigned int numberOfOuterVertices, const unsigned int numberOfOuterTriangles,
                             bool* inside)
{
    unsigned int innerVertexId = threadIdx.x + blockIdx.x * blockDim.x;

    if(innerVertexId < numberOfInnerVertices){

        unsigned int numberOfIntersections = 0;

        CudaRay ray(innerVertices[innerVertexId], glm::vec3(1, 0, 0));

        for(int outerTriangleIndex=0; outerTriangleIndex<numberOfOuterTriangles; outerTriangleIndex++){
            Triangle outerTriangle = outerTriangles[outerTriangleIndex];
            Vertex v0 = outerVertices[outerTriangle.vertexIndex0];
            Vertex v1 = outerVertices[outerTriangle.vertexIndex1];
            Vertex v2 = outerVertices[outerTriangle.vertexIndex2];
            if(ray.intersectsTriangle(v0,v1,v2)){
                numberOfIntersections++;
            }
        }
        if(numberOfIntersections % 2 == 0){
            *inside = false;
            return;
        }
    }

}

bool CudaWorldSpaceMesh::rayTriangleInside(const CudaWorldSpaceMesh &other) const {

    unsigned blockSize = BLOCK_SIZE;
    unsigned int nBlocks= this->numberOfVertices / blockSize + (this->numberOfVertices % blockSize == 0 ? 0 : 1);

    bool inside = true;
    hipMemcpyAsync(&flags[0], &inside, sizeof(bool), hipMemcpyHostToDevice, *this->cudaStream);

    rayTriangleInsideKernel <<< nBlocks, blockSize, 0, *this->cudaStream >>>
    (this->triangles, this->worldSpaceVertices, this->numberOfVertices, this->numberOfTriangles,
     other.triangles, other.worldSpaceVertices, other.numberOfVertices, other.numberOfTriangles, &flags[0]);

    hipMemcpyAsync(&inside, &flags[0], sizeof(bool), hipMemcpyDeviceToHost, *this->cudaStream);

    hipStreamSynchronize(*this->cudaStream);

    return inside;
}

__global__
void triangleTriangleIntersectKernel(const Triangle* innerTriangles, const Vertex* innerVertices, const unsigned numberOfInnerVertices, const unsigned int numberOfInnerTriangles,
                                     const Triangle* outerTriangles, const Vertex* outerVertices, const unsigned int numberOfOuterVertices, const unsigned int numberOfOuterTriangles,
                                     bool* intersects)
{
    unsigned int threadId = threadIdx.x + blockIdx.x * blockDim.x;

    unsigned int innerTriangleId = threadId;

//    unsigned int innerTriangleId = threadId/numberOfOuterTriangles;
//    unsigned int outerTriangleId = threadId%numberOfOuterTriangles;

    if(innerTriangleId < numberOfInnerTriangles){
        Triangle innerTriangle = innerTriangles[innerTriangleId];
        Vertex innerVertex0 = innerVertices[innerTriangle.vertexIndex0];  // It's worth it transforming all vertices in advance
        Vertex innerVertex1 = innerVertices[innerTriangle.vertexIndex1];
        Vertex innerVertex2 = innerVertices[innerTriangle.vertexIndex2];

        for(int outerTriangleIndex=0; outerTriangleIndex<numberOfOuterTriangles; outerTriangleIndex++){
            Triangle outerTriangle = outerTriangles[outerTriangleIndex];
            Vertex outerVertex0 = outerVertices[outerTriangle.vertexIndex0];
            Vertex outerVertex1 = outerVertices[outerTriangle.vertexIndex1];
            Vertex outerVertex2 = outerVertices[outerTriangle.vertexIndex2];
            bool test = CudaIntersection::NoDivTriTriIsect(glm::value_ptr(innerVertex0),
                                                           glm::value_ptr(innerVertex1),
                                                           glm::value_ptr(innerVertex2),
                                                           glm::value_ptr(outerVertex0),
                                                           glm::value_ptr(outerVertex1),
                                                           glm::value_ptr(outerVertex2));
            if (test) {
                *intersects = true;
            }
        }
    }
}

bool CudaWorldSpaceMesh::triangleTriangleIntersects(const CudaWorldSpaceMesh &other) const {

    unsigned blockSize = BLOCK_SIZE;
//    unsigned int nThreads = this->numberOfTriangles * other.numberOfTriangles;
    unsigned int nThreads = this->numberOfTriangles;
    unsigned int nBlocks= nThreads / blockSize + (nThreads % blockSize == 0 ? 0 : 1);

    bool intersects = false;

    hipMemcpyAsync(&flags[1], &intersects, sizeof(bool), hipMemcpyHostToDevice, *this->cudaStream);

    triangleTriangleIntersectKernel <<< nBlocks, blockSize, 0, *this->cudaStream >>>
            (this->triangles, this->worldSpaceVertices, this->numberOfVertices, this->numberOfTriangles,
             other.triangles, other.worldSpaceVertices, other.numberOfVertices, other.numberOfTriangles, &flags[1]);

    hipMemcpyAsync(&intersects, &flags[1], sizeof(bool), hipMemcpyDeviceToHost, *this->cudaStream);

    hipStreamSynchronize(*this->cudaStream);

    return intersects;
}